#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define N 128

__global__ void calc_freq(int *freq, int file_size, char *buffer, int total_threads){
	int temp[N];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;  

	// Initialize temp with value 0
	for (int i = 0; i < N; i++){
		temp[i] = 0;
	}

	// Do the calculation
	for(int i = idx; i < file_size; i += total_threads) {
     	temp[buffer[i]]++;	
	}

	// Add the results from the threads to the blocks
	for(int i = 0; i < N; i++){
		atomicAdd(&freq[i], temp[i]);
	}

}

int main (int argc, char *argv[]) {

	int *freq_host, *sum_device;
	char * buffer_device;

	// Error handling for arguments
	if (argc != 4) {
		printf ("Usage : %s <file_name> <blocks> <threads_per_block>\n", argv[0]);
		return 1;
	}
	
	// Open file
	char *filename = argv[1];
	FILE *pFile = fopen ( filename , "rb" );
	if (pFile==NULL) {printf ("File error\n"); return 2;}

	// Obtain file size:
	fseek (pFile , 0 , SEEK_END);
	long file_size = ftell (pFile);
	rewind (pFile);
	printf("file size is %ld\n", file_size);
	
	// Allocate memory to contain the file:
	char *buffer = (char*) malloc (sizeof(char)*file_size);
	if (buffer == NULL) {printf ("Memory error\n"); return 3;}

	// Copy the file into the buffer:
	size_t result = fread (buffer,1,file_size,pFile);
	if (result != file_size) {printf ("Reading error\n"); return 4;} 
	
	// Create space for required variables
	freq_host = (int*) malloc(N * sizeof(int));
	hipMalloc((void **) &sum_device, N * sizeof(int));
	hipMalloc((void **) &buffer_device, file_size * sizeof(char));

	// Initialize freq_host with 0
	for (int i = 0; i < N; i++){
		freq_host[i]=0;
	}

	// Copy required values to device variables
	hipMemcpy(sum_device, freq_host, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(buffer_device, buffer, file_size * sizeof(char), hipMemcpyHostToDevice);

	// get blocks, threads per block and total amount of threads
	int threads = strtol(argv[3], NULL, 10);
	int blocks = (strtol(argv[2], NULL, 10) + threads - 1) / threads;
	int total_threads = blocks * threads;
	
	// Begin parallel code
	calc_freq<<<blocks, threads>>>(sum_device, file_size, buffer_device, total_threads);

	// Copy result from device to host
	hipMemcpy(freq_host, sum_device, N * sizeof(int), hipMemcpyDeviceToHost);

	// Display results
	for (int j = 0; j < N; j++){
		printf("%c = %d\n", j, freq_host[j]);
	}	

	// Free variables
	fclose (pFile);
	free (buffer);
	hipFree(freq_host);
	hipFree(sum_device);

	return 0;
}