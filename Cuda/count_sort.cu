
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void count_sort(int *x, int *y, int size){

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < size){

    int count = 0;

    for(int j = 0; j < size; j++){

      if (x[j] < x[idx])
        count++;
      else if (x[j] == x[idx] && j < idx)
        count++;

    }

    y[count] = x[idx];
  }

}

void generate_array(int size, int* x_host, int* y_host, int limit) {

  //Initializion for rand();
  srand(23);

  for (int i = 0; i < size; i++){
    *(x_host + i) = rand() % limit;
    *(y_host + i) = 0;
  }
}

int main(int argc, char *argv[]) {

  int *x_host, *y_host, *x_device, *y_device;

  if (argc != 3) {
    printf ("Usage : %s <array_size> <Threads_per_block>\n", argv[0]);
    return 1;
  }

  int size = strtol(argv[1], NULL, 10);

  int threads = strtol(argv[2], NULL, 10);
  int blocks = (size + threads - 1) / threads;

  // Allocate memory as arrays on host
  x_host = (int*) malloc(size * sizeof(int));
  y_host = (int*) malloc(size * sizeof(int));

  // Allocate memory as arrays on device
  hipMalloc( &x_device, size * sizeof(int));
  hipMalloc((void **) &y_device, size * sizeof(int));

  generate_array(size, x_host, y_host, 100);
  // for (int i=0; i<size; i++){
  //   x_host[i] = size - i;
  //   y_host[i] = 0;
  // }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Copy data to device
  hipMemcpy(x_device, x_host, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(y_device, y_host, size * sizeof(int), hipMemcpyHostToDevice);
  
  hipEventRecord(start);

  // Do the calculations
  count_sort<<<blocks, threads>>>(x_device, y_device, size);
  
  hipEventRecord(stop);

  // Get data from device to host
  hipMemcpy(y_host, y_device, size * sizeof(int), hipMemcpyDeviceToHost);
  
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  for (int i=0; i<size; i++){
    printf("%d\n", y_host[i]);
  }
  
  printf("GPU time (s): %f\n", milliseconds/1000);

  // Free variables
  free(x_host);
  free(y_host);
  hipFree(x_device);
  hipFree(y_device);

  return 0;
}
