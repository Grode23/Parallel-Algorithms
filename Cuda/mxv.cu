
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Device code
__global__ void calc(float *result, float *b, float *a, int size){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < size){
		
		float temp;
		
		for (int j = 0; j < size; j++){
			temp = *(a + j + (idx * size)) * (*(b + j));
			atomicAdd(&result[idx], temp);
		}
	}
}

int main (int argc, char *argv[]) {
	float *a, *b, *c;
	float *b_device, *c_device, *a_device;
	int size;

	if (argc != 3) {
		printf ("Usage : %s <matrix size> <threads>\n", argv[0]);
		exit(1);
	}

	// Get size from agruments
	size = strtol(argv[1], NULL, 10);
	
	// Allocate the matrices
	a = (float*) malloc (size * size * sizeof(float));
	b = (float*) malloc (size * sizeof(float));
	c = (float*) malloc (size * sizeof(float));
	hipMalloc((void **) &c_device, size * sizeof(float));
	hipMalloc((void **) &b_device, size * sizeof(float));
	hipMalloc((void **) &a_device, size * size * sizeof(float));

	// Assign values to the B and C matrices
	srand ( time ( NULL));

	for (int i = 0; i < size; i++ ) 
		for (int j = 0; j < size; j++ )
			*(a + j + (i * size)) = ( float ) rand() / (RAND_MAX * 2.0 - 1.0);

		for (int i = 0; i < size; i++ ) {
	    b[i] = ( float ) rand() / (RAND_MAX * 2.0 - 1.0);
	    c[i] = 0.0;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(c_device, c, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(a_device, a, size * size * sizeof(float), hipMemcpyHostToDevice);

	// User can choose only the number of threads per blocks 
	// The blocks will be calculated automatically
	int threads = strtol(argv[2], NULL, 10);
	int blocks = (size + threads - 1) / threads;

	hipEventRecord(start);

	calc<<<blocks, threads>>>(c_device, b_device, a_device, size);
	
	hipEventRecord(stop);

	hipMemcpy(c, c_device, size * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	for (int i = 0; i < size; i++ ) {
		printf("\t %1.3f ", b[i]);
		printf("\t %1.3f \n", c[i]);
	}
	
	printf("GPU time (ms): %f\n", milliseconds);

	free(a);
	free(b);
	free(c);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

	return 0;

}